#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cmath>
#include <stdlib.h>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "cublas_utils.h"

using namespace std;
using data_type = float;


__global__ void sdot2_4(data_type *a, data_type *b, data_type *c, int n){

    // Define variables.
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    data_type temp;
    temp = 0;
    // Define shared memories.
    __shared__ data_type s_data[1024];
    unsigned int tid = threadIdx.x;
    // Multiplication of data in the index.
    for (int i = index; i < n; i += stride){
        temp += ( a[i] * b[i] );
    }
    // Assign value to shared memory.
    s_data[tid] = temp;
    __syncthreads();
    // Add up products.
    for (int s = blockDim.x / 4; s > 0 ; s >>= 2){
        if ((tid < s)) {
            temp = s_data[tid];
            temp += s_data[tid + s];
            temp += s_data[tid + (s << 1)];
            temp += s_data[tid + (3 * s)];
            s_data[tid] = temp;
        }
        __syncthreads();
    }
    if(tid == 0){
        atomicAdd( c , s_data[0] );
    }
}


__global__ void sdot2_2(data_type *a, data_type *b, data_type *c, int n){

    // Define variables.
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    data_type temp;
    temp = 0;
    // Define shared memories.
    __shared__ data_type s_data[1024];
    unsigned int tid = threadIdx.x;
    // Multiplication of data in the index.
    for (int i = index; i < n; i += stride){
        temp += ( a[i] * b[i] );
    }
    // Assign value to shared memory.
    s_data[tid] = temp;
    __syncthreads();
    // Add up products.
    for (int s = blockDim.x / 4; s > 0 ; s >>= 2){
        if ((tid < s)) {
            temp = s_data[tid];
            temp += s_data[tid + s];
            temp += s_data[tid + (s << 1)];
            temp += s_data[tid + (3 * s)];
            s_data[tid] = temp;
        }
        __syncthreads();
    }
    s_data[0] += s_data[1];
    if(tid == 0){
        atomicAdd( c , s_data[0] );
    }
}


__global__ void sdot1(data_type *a, data_type *b, data_type *c, int n){
    // Define variables.
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    data_type temp = 0.0;
    __shared__ data_type s_data[1024];
    // __shared__ data_type s_data[(int)gridDim.x];
    unsigned int tid = threadIdx.x;
    // Multiplication of data in an index.
    for (int i = index; i < n; i += stride){
        temp += ( a[i] * b[i] );
    }
    s_data[tid] = temp;
    __syncthreads();
    for (int s = blockDim.x / 2; s > 0 ; s >>= 1){
        if ((tid < s) && (index + s < n))
        {
            s_data[tid] += s_data[tid + s];
        }
        __syncthreads();
    }
    if(tid == 0){
        atomicAdd( c , s_data[0] );
    }
}


int main(int argc, char *argv[]){

    // Define random seed
    srand(0);

    // Parameter assignment.
    int N, nBytes;
    int kernel_num, block_size;
    data_type *A, *B, *C;

    int start_n = 100;
    int step_n = 200;
    int repeat_n = 20;
    int end_n = start_n + (repeat_n * step_n);

    // Set parameter
    printf("Kernel Number (0 for CuBLAS): ");
    scanf("%d", &kernel_num);
    printf("Block Size: ");
    scanf("%d", &block_size);
    block_size = 256;

    for (int repeat_t = start_n; repeat_t <= end_n; repeat_t += step_n){

        // Initialize cuda event
        hipEvent_t start, end;
        hipEventCreate(&start);
        hipEventCreate(&end);

        // Memory size assignment
        N = repeat_t * repeat_t;
        nBytes = N * sizeof(data_type);

        // Allocate memory of the host to store data.
        A = (data_type*)malloc(nBytes);
        B = (data_type*)malloc(nBytes);
        C = (data_type*)malloc(sizeof(data_type));

        // Assign data to the variable.
        for (int i = 0; i < N; ++i)
        {
            A[i] = (float)((rand()%10000)/100);
            B[i] = (float)((rand()%10000)/100);
        }
        *C = 0;
        printf("------------------------\n");
        printf("Length of Vector = %d\n", N);  

        // Allocate memory of the device to store data.
        data_type *d_A, *d_B, *d_C;
        hipMalloc((void**)&d_A, nBytes);
        hipMalloc((void**)&d_B, nBytes);
        hipMalloc((void**)&d_C, sizeof(data_type));

        // Copy data from host to device.
        hipMemcpy((void*)d_A, (void*)A, nBytes, hipMemcpyHostToDevice);
        hipMemcpy((void*)d_B, (void*)B, nBytes, hipMemcpyHostToDevice);
        hipMemcpy((void*)d_C, (void*)C, sizeof(data_type), hipMemcpyHostToDevice);

        // Define the configuration.
        dim3 blockSize(block_size);
        dim3 gridSize(min(1024,(N + blockSize.x - 1) / blockSize.x));



        if (kernel_num == 2){
            float isInt = log((float)block_size)/log(4);
            printf("%f", isInt);
            if (isInt == (float)(int)isInt){
                //Start timer.
                hipEventRecord(start);

                // Run the kernel.
                sdot2_4 <<< gridSize, blockSize >>>(d_A, d_B, d_C, N);

                // End timer, Calculate performance.
                hipEventRecord(end);
            }
            else{
                //Start timer.
                hipEventRecord(start);

                // Run the kernel.
                sdot2_2 <<< gridSize, blockSize >>>(d_A, d_B, d_C, N);

                // End timer, Calculate performance.
                hipEventRecord(end);
            }
        }
        else if(kernel_num == 1){
            //Start timer.
            hipEventRecord(start);

            // Run the kernel.
            sdot1 <<< gridSize, blockSize >>>(d_A, d_B, d_C, N);

            // End timer, Calculate performance.
            hipEventRecord(end);
        }
        else if(kernel_num == 0){
            hipblasHandle_t handle;
            hipblasCreate(&handle); 
            //Start timer.
            hipEventRecord(start);
            // Run the kernel.    
            hipblasSdot(handle, N, d_A, 1, d_B, 1, d_C);
            // End timer, Calculate performance.
            hipEventRecord(end);
        }




        hipEventSynchronize(start);
        hipEventSynchronize(end);
        float elapsedtime = 0.0;
        double flops;
        hipEventElapsedTime(&elapsedtime, start, end);
        flops = (double)((2 * N) - 1) / (double)elapsedtime;
        flops /= 1000000.0;

        // Copy the result from device to host.
        hipMemcpy((void*)C, (void*)d_C, sizeof(float), hipMemcpyDeviceToHost);


        printf("Time spent: %f ms\n", elapsedtime);
        printf("Performance: %f GFLOPS", flops);
        printf("\n========================\n\n");



        // Release memory on device.
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);


        // Release memory on host.
        free(A);
        free(B);
        free(C);
        


}










}